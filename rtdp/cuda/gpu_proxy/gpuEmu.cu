#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------
// This program will:
//  1. Listen to the imcoming traffic from a TCP port via ZMQ.
//  2. Copy the incoming data to GPU and do some fake calculation on GPU.
//  3. Copy the results from GPU to CPU and send them out via ZMQ.
//
//---------------------------------------------------------------------------

#include <iostream>
#include <thread>
#include <string>
#include <chrono>
#include <vector>
#include <cmath>
#include <unistd.h> // For getpid()

#include <zmq.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>


// ZMQ settings
constexpr const int ZMQ_IN_PORT = 55555;
constexpr const int ZMQ_OUT_PORT = 55556;
constexpr const char* ZMQ_RECV_ADDR = "tcp://*:55555";
constexpr const char* ZMQ_SEND_ADDR = "tcp://*:55556";

/// TODO: @xmei, Design the SQL Rate Logger
// #include "SQLiteRateLogger.h"  // Check Podio2tcp

// Global SQlite Rate logger
/// NOTE: data schema for the receiver DB
// sqlite> .schema rate_logs
// CREATE TABLE rate_logs (
//     id INTEGER PRIMARY KEY AUTOINCREMENT,
//     timestamp_utc_ms INTEGER,
//     pid STRING,
//     rateHz_recv_period REAL,
//     rateMbps_recv_period REAL
// );
// SQLiteRateLogger rate_logger;
// std::string RATE_DB_COLUMNS = "timestamp_utc_ms, pid, "
//                             "rateHz_recv_period, "
//                             "rateMbps_recv_period";


//.........................................................................
// Matrix multiplication setup

constexpr int MATRIX_IN_COLUMN_WIDTH = 2048;
constexpr float MATRIX_OUT_REDUCE_RATE = 0.5; // Define reduction rate

// CUDA error check
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(x) << std::endl; \
    exit(EXIT_FAILURE); }} while(0)

// GPU kernel function
__global__ void generateRandomMatrix(float* d_rand, int rows, int cols, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = rows * cols;
    if (idx < totalElements) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        d_rand[idx] = hiprand_uniform(&state);
    }
}

void matrixProcess(float* d_A, float* d_B, int rows, int cols) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f, beta = 0.0f;
    int reducedCols = cols * MATRIX_OUT_REDUCE_RATE;
    /// TODO: add comments for hipblasSgemm and make reduceCols from input param
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, reducedCols, rows, cols, &alpha, d_A, cols, d_B, cols, &beta, d_A, reducedCols);
    hipblasDestroy(handle);
}

void processMatrixComputation(float* d_A, float* d_B, int rows, int cols) {
    /// TODO: check the option for selecting GPU TC vs FP
    matrixProcess(d_A, d_B, rows, cols);
}
//.........................................................................


//.........................................................................
// Based on the podio2tcp application.
class CommandLineOptions {
public:
    std::string inputFilename;
    std::string outfile;
    int recv_port = ZMQ_IN_PORT;
    int send_port = ZMQ_OUT_PORT;
    double rate = 0.0; // Unset
    std::string sqliteFilename;      // SQL file parameter
    std::string ipAddress = "localhost";

    static CommandLineOptions Parse(int argc, char* argv[]) {
        CommandLineOptions options;
        /// TODO: add a mode for using FP or TC on GPUs
        for (int i = 1; i < argc; ++i) {
            std::string arg = argv[i];
            if (arg == "-h" || arg == "--help") {
                PrintUsage();
                exit(0);
            } else if (arg == "--in-port") {
                if (i + 1 < argc) {
                    options.recv_port = std::stoi(argv[++i]);
                }
            } else if (arg == "--out-port") {
                if (i + 1 < argc) {
                    options.send_port = std::stoi(argv[++i]);
                }
            } else if (arg == "-s" || arg == "--sqlfile") {
                if (i + 1 < argc) {
                    options.sqliteFilename = argv[++i];
                }
            }
        }

        return options;
    }

    static void PrintUsage() {
        std::cout << "\n" 
                  << "Usage: gpu_emu [--in-port port] [--out-port port]\n"
                  << "\n"
                  << "-h, --help   Print this help statement\n"
                  << "    --in-port  <port> Set ZMQ port to listen on (default is 55555)\n"
                  << "    --out-port <port> Set ZMQ port to push to (default is 55556)\n"
                  << "-s, --sqlfile <file> Specify the SQL rate logger file\n"
                  << "\n"
                  << "This is xxxx\n."
                  << "xxx\n"
                  << "xxx."
                  << "\n"
                  << "If --sqlfile is used, it specifies the SQLite database output.\n"
                  << "\n";
    }
};
//.........................................................................


//.........................................................................
// The monitoring thread
void monitorTraffic(size_t* inBytes, size_t* outBytes) {
    using namespace std::chrono;
    while (true) {
        size_t prevIn = *inBytes, prevOut = *outBytes;
        std::this_thread::sleep_for(seconds(1));
        size_t curIn = *inBytes, curOut = *outBytes;
        std::cout << "Incoming: " << ((curIn - prevIn) * 8.0 / 1e6) << " Mbps, "
                  << "Outgoing: " << ((curOut - prevOut) * 8.0 / 1e6) << " Mbps" << std::endl;
    }
}
//.........................................................................


//.........................................................................
// Main
int main(int narg, char *argv[]){

    // Parse command options (will print help and exit if help is asked for)
    CommandLineOptions options = CommandLineOptions::Parse(narg, argv);

    //............................................
    // Setup network communication via zmq
    zmq::context_t context(1);

    // Receiving socket
    zmq::socket_t receiver(context, ZMQ_PULL);
    // Taken from podio2tcp: set High Water Mark for maximum number of messages to queue before stalling
    receiver.set(zmq::sockopt::rcvhwm, 10);
    try {
        receiver.bind(ZMQ_RECV_ADDR);
        std::cout << "ZeroMQ listening at: " << ZMQ_RECV_ADDR << "\n";
    }  catch (const zmq::error_t& e) {
        std::cout << "Error: Failed to bind to the listening address [" << ZMQ_RECV_ADDR << "]:" << e.what() << "\n";
        return 1;
    }

    // Sending socket
    zmq::socket_t sender(context, ZMQ_PUSH);
    sender.set(zmq::sockopt::rcvhwm, 10);
    try {
        sender.bind(ZMQ_SEND_ADDR);
        std::cout << "ZeroMQ listening at: " << ZMQ_SEND_ADDR << "\n";
    }  catch (const zmq::error_t& e) {
        std::cout << "Error: Failed to bind to the sending address [" << ZMQ_SEND_ADDR << "]:" << e.what() << "\n";
        return 1;
    }
    //............................................

    //............................................
    // Monitoring thread
    // size_t inBytes = 0, outBytes = 0;
    // std::thread monitor(monitorTraffic, &inBytes, &outBytes);
    // monitor.detach();
    //............................................

    //............................................
    // SQL logger setup
    // if (!options.sqliteFilename.empty() && !rate_logger.openDB(options.sqliteFilename)) {
    //     std::cerr << "Failed to open database: " << options.sqliteFilename << std::endl;
    //     return 1;
    // }
    //............................................


    std::cout << "\nWaiting for data ..." << std::endl;


    auto last_time = std::chrono::high_resolution_clock::now();
    while (true) {
        zmq::message_t recv_buffer;
        auto res = receiver.recv(recv_buffer, zmq::recv_flags::none);
        if (!res) {
            std::cerr << "Error: ZeroMQ receive failed!" << std::endl;
        } else {
            std::cout << "Received [" << res.value() << "] bytes from ZeroMQ socket." << std::endl;
        }
        
        size_t curr_inBytes = recv_buffer.size();
        if( curr_inBytes == 0 ) { 
            std::cout << "  (skipping empty buffer)" << std::endl;
            continue;
        }

        float *d_in, *d_rand;
        // Setup the input matrix A [rows * in_columns (default as 2048)] on the CPU side
        int totalElements = curr_inBytes / sizeof(float);
        int rows = (totalElements + MATRIX_IN_COLUMN_WIDTH - 1) / MATRIX_IN_COLUMN_WIDTH;
        int in_cols = MATRIX_IN_COLUMN_WIDTH;

        std::vector<float> h_in(rows * in_cols, 0);
        memcpy(h_in.data(), recv_buffer.data(), curr_inBytes);

        // Copy input matrix to GPU
        CUDA_CALL(hipMalloc(&d_in, rows * in_cols * sizeof(float)));
        CUDA_CALL(hipMemcpy(d_in, h_in.data(), rows * in_cols * sizeof(float), hipMemcpyHostToDevice));

        // Set the random matrix d_rand on the GPU. d_rand has @var MATRIX_IN_COLUMN_WIDTH rows.
        int out_cols = std::ceil(MATRIX_IN_COLUMN_WIDTH * MATRIX_OUT_REDUCE_RATE);
        int rand_elements = MATRIX_IN_COLUMN_WIDTH * out_cols;
        CUDA_CALL(hipMalloc(&d_rand, rand_elements * sizeof(float)));

        int threadsPerBlock = 256;
        int numBlocks = (rand_elements + threadsPerBlock - 1) / threadsPerBlock;
        generateRandomMatrix<<<numBlocks, threadsPerBlock>>>(d_rand, MATRIX_IN_COLUMN_WIDTH, out_cols, time(NULL));
        CUDA_CALL(hipDeviceSynchronize());

        // Do the matrixMul with cublas function call
        processMatrixComputation(d_in, d_rand, rows, in_cols);

        // Copy the result matrix back to host
        std::vector<float> h_out(rows * out_cols);
        CUDA_CALL(hipMemcpy(h_out.data(), d_in, rows * out_cols * sizeof(float), hipMemcpyDeviceToHost));

        CUDA_CALL(hipFree(d_in));
        CUDA_CALL(hipFree(d_rand));

        zmq::message_t reply(rows * out_cols * sizeof(float));
        memcpy(reply.data(), h_out.data(), rows * out_cols * sizeof(float));
        res = sender.send(reply, zmq::send_flags::none);
        if (!res) {
            std::cerr << "Error: ZeroMQ send failed!" << std::endl;
        } else {
            std::cout << "Sent [" << res.value() << "] bytes via ZeroMQ socket." << std::endl;
        }
        // outBytes += reply.size();

        // Print statistic
        /// TODO: move to a monitoring thread instead.
        auto now = std::chrono::high_resolution_clock::now();
        // auto duration = std::chrono::duration<double>(now - last_time).count();
        // auto rateMbps = curr_inBytes / duration * 8.0 / 1.0E6;
        // auto savePrecision = std::cout.precision();
        // std::cout << "  INCOMING data rate: " << std::fixed << std::setprecision(3)  << rateMbps << " (Mbps)" << std::endl;
        // std::cout.precision(savePrecision);

        // Log to SQLite DB
        // auto utc_timestamp_in_ms =
        //     std::chrono::duration_cast<std::chrono::milliseconds>(now.time_since_epoch()).count();
        // std::string pid_str = std::to_string(getpid());
        // std::ostringstream values;
        // values << std::to_string(utc_timestamp_in_ms) << ", "
        //     << pid_str << ", "
        //     << std::fixed << std::setprecision(3)  // Ensure consistent floating-point precision
        //     << rateHz << ", "
        //     << rateMbps;
        // if (!rate_logger.insertRateLog(RATE_DB_COLUMNS, values.str())) {
        //     std::cerr << "Failed to insert record into the database." << std::endl;
        // }

        last_time = now;

        // std::this_thread::sleep_for(std::chrono::seconds(1));
    }
    // Close SQLite3 DB
    // rate_logger.closeDB();

    return 0;
}