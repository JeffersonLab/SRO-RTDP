#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------
// This program will:
//  1. Listen to the imcoming traffic from a TCP port via ZMQ.
//  2. Copy the incoming data to GPU and do some fake calculation on GPU.
//  3. Copy the results from GPU to CPU and send them out via ZMQ.
//
//---------------------------------------------------------------------------

#include <iostream>
#include <thread>
#include <string>
#include <chrono>
#include <vector>
#include <cmath>
#include <atomic>
#include <unistd.h> // For getpid()

#include <zmq.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>


// ZMQ settings
constexpr const int ZMQ_IN_PORT = 55555;
constexpr const int ZMQ_OUT_PORT = 55556;

/// TODO: @xmei, Design the SQL Rate Logger
// #include "SQLiteRateLogger.h"  // Check Podio2tcp

// Global SQlite Rate logger
/// NOTE: data schema for the receiver DB
// sqlite> .schema rate_logs
// CREATE TABLE rate_logs (
//     id INTEGER PRIMARY KEY AUTOINCREMENT,
//     timestamp_utc_ms INTEGER,
//     pid STRING,
//     rateHz_recv_period REAL,
//     rateMbps_recv_period REAL
// );
// SQLiteRateLogger rate_logger;
// std::string RATE_DB_COLUMNS = "timestamp_utc_ms, pid, "
//                             "rateHz_recv_period, "
//                             "rateMbps_recv_period";


//.........................................................................
// Matrix multiplication setup

constexpr int MATRIX_IN_COLUMN_WIDTH = 2048;
constexpr float MATRIX_OUT_REDUCE_RATE = 0.5; // Define reduction rate

// CUDA error check
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(x) << std::endl; \
    exit(EXIT_FAILURE); }} while(0)

// CUBLAS error check
#define CUBLAS_CALL(call) \
do { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "CUBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// GPU kernel function
__global__ void generateRandomMatrix(float* d_rand, int rows, int cols, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = rows * cols;
    if (idx < totalElements) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        d_rand[idx] = hiprand_uniform(&state);
    }
}

// Excute on GPU
void matrixProcess(float* d_A, float* d_B, float* d_C, int rows, int in_cols, int out_cols) {
    hipblasHandle_t handle;
    CUBLAS_CALL(hipblasCreate(&handle));
    const float alpha = 1.0f, beta = 0.0f;
    
    // Perform matrix multiplication: C = A * B
    CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            out_cols, rows, in_cols, 
                            &alpha, d_B, out_cols, 
                            d_A, in_cols, 
                            &beta, d_C, out_cols));
    
    CUBLAS_CALL(hipblasDestroy(handle));
}

// Execute on CPU.
void cpuMatrixMultiply(
    const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& C,
    int rows, int in_cols, int out_cols) {
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < out_cols; ++j) {
                float sum = 0.0f;
                for (int k = 0; k < in_cols; ++k) {
                    sum += A[i * in_cols + k] * B[k * out_cols + j];
                }
                C[i * out_cols + j] = sum;
            }
        }
}
//.........................................................................


//.........................................................................
// Based on the podio2tcp application.
class CommandLineOptions {
public:
    int recv_port = ZMQ_IN_PORT;
    int send_port = ZMQ_OUT_PORT;
    double rate = MATRIX_OUT_REDUCE_RATE;
    int width = MATRIX_IN_COLUMN_WIDTH; // Default matrix column size

    std::string out_ip = "127.0.0.1";  // Default out IP is local
    bool useTensorCores = false;
    bool verbose = false;

    std::string sqliteFilename;      // SQL file parameter

    static CommandLineOptions Parse(int argc, char* argv[]) {
        CommandLineOptions options;
        for (int i = 1; i < argc; ++i) {
            std::string arg = argv[i];
            if (arg == "-h" || arg == "--help") {
                PrintUsage();
                exit(0);
            } else if (arg == "--in-port" && i + 1 < argc) {
                options.recv_port = std::stoi(argv[++i]);
            } else if (arg == "-a" || arg == "--out-ip") {
                if (i + 1 < argc) {
                    options.out_ip = argv[++i];
                }
            } else if (arg == "--out-port" && i + 1 < argc) {
                options.send_port = std::stoi(argv[++i]);
            } else if (arg == "-r" || arg == "--rate") {
                if (i + 1 < argc) {
                    options.rate = std::stod(argv[++i]);
                }
            } else if (arg == "-w" || arg == "--width") {
                if (i + 1 < argc) {
                    options.width = std::stoi(argv[++i]);
                }
            } else if (arg == "-s" || arg == "--sqlfile") {
                if (i + 1 < argc) {
                    options.sqliteFilename = argv[++i];
                }
            } else if (arg == "--tc") {
                options.useTensorCores = true;
            } else if (arg == "-v" || arg == "--verbose") {
                options.verbose = true;
            }
        }
        return options;
    }


    static void PrintUsage() {
        std::cout << "\n" 
                  << "Usage: gpu_emu [--in-port] [-a|--out-ip] [--out-port]\n"
                  << "\n"
                  << "-h, --help     Print this help statement\n"
                  << "    --in-port  <port> Set ZMQ port to listen on (default is 55555)\n"
                  << "-a, --out-ip   <IP_Address> The IP address ZMQ push to (default is localhost)\n"
                  << "    --out-port <port> Set ZMQ port to push to (default is 55556)\n"
                  << "-r, --rate     Control the ratio of output/input volume (default is 0.5)\n"
                  << "-w, --width    Set the GPU input matrix column size (default is 2048)\n"
                //   << "-s, --sqlfile  <file> Specify the SQL rate logger file\n"
                //   << "    --tc       Use GPU Tensor Cores instead of FP units\n"
                  << "-v, --verbose    Enable the verbose mode (default is false)\n"

                  << "\n"
                  << "This is a GPU Proxy\n"
                  << "It takes input from a ZMQ PULL port and builds a matrix, sends to GPU and do matrix\n"
                  << "multiplication on the GPU. After that, it copies the result back to CPU and PUSH to\n"
                  << "another ZMQ IP & port.\n"
                  << "\n"
                //   << "If --sqlfile is used, it specifies a SQLite rate logger.\n"
                  << "\n";
    }
};
//.........................................................................


//.........................................................................
// The monitoring thread
std::atomic<size_t> totalInBytes{0};
std::atomic<size_t> totalOutBytes{0};

void monitorTraffic(std::atomic<size_t>* inBytes, std::atomic<size_t>* outBytes) {
    using namespace std::chrono;
    while (true) {
        size_t prevIn = inBytes->load();
        size_t prevOut = outBytes->load();
        std::this_thread::sleep_for(seconds(2));
        size_t curIn = inBytes->load();
        size_t curOut = outBytes->load();

        double inRate_MBps = (curIn - prevIn) / (1000.0 * 1000.0) / 2.0;
        double outRate_MBps = (curOut - prevOut) / (1000.0 * 1000.0) / 2.0;

        std::cout << "[Monitor] Incoming: [" << inRate_MBps << " MB/s], "
                  << "Outgoing: [" << outRate_MBps << " MB/s]" << std::endl;
    }
}
//.........................................................................


//.........................................................................
// Main
int main(int narg, char *argv[]){

    // Parse command options (will print help and exit if help is asked for)
    CommandLineOptions options = CommandLineOptions::Parse(narg, argv);

    // Enable the verbose mode if the cmd flag is provided
    bool verbose_mode = options.verbose;

    //............................................
    // Setup network communication via zmq
    zmq::context_t context(1);

    // Receiving socket
    zmq::socket_t receiver(context, ZMQ_PULL);
    // Taken from podio2tcp: set High Water Mark for maximum number of messages to queue before stalling
    receiver.set(zmq::sockopt::rcvhwm, 10000);
    std::string recv_addr = "tcp://*:" + std::to_string(options.recv_port);
    try {
        // Listen to localhost only
        receiver.bind(recv_addr.c_str());
        std::cout << "RECV - ZeroMQ pulling from: " << recv_addr << "\n";
    }  catch (const zmq::error_t& e) {
        std::cout << "Error: Failed to bind to the receiving address [" << recv_addr << "]:" << e.what() << "\n";
        return 1;
    }

    // Sending socket
    zmq::socket_t sender(context, ZMQ_PUSH);
    // Set High Water Mark for maximum number of messages to queue before stalling
    sender.set(zmq::sockopt::sndhwm, 10000);
    std::string send_addr = "tcp://" + options.out_ip + ":" + std::to_string(options.send_port);
    try {
        sender.connect(send_addr.c_str());  // NOTE: connect() not bind()!!!!
        std::cout << "SEND - ZeroMQ pushing to: " << send_addr << "\n";
    }  catch (const zmq::error_t& e) {
        std::cout << "Error: Failed to connect to the sending address [" << send_addr << "]:" << e.what() << "\n";
        return 1;
    }
    //............................................

    std::cout << "\nWaiting for data ...\n" << std::endl;

    std::thread monitor_thread(monitorTraffic, &totalInBytes, &totalOutBytes);
    monitor_thread.detach();   // Start the rate monitoring thread

    while (true) {
        zmq::message_t recv_buffer;
        auto res = receiver.recv(recv_buffer, zmq::recv_flags::none);
        if (!res) {
            std::cerr << "Error: ZeroMQ receive failed!" << std::endl;
        }

        if (verbose_mode) {
            std::cout << "Received [" << res.value() << "] bytes from ZeroMQ socket." << std::endl;
        }
        
        size_t curr_inBytes = recv_buffer.size();
        totalInBytes += curr_inBytes;
        if( curr_inBytes == 0 ) { 
            if (verbose_mode) {
                std::cout << "  (skipping empty buffer)" << std::endl;
            }
            continue;
        }

        float *d_in, *d_rand, *d_out;
        // Setup the input matrix A [rows * in_columns (default as 2048)] on the CPU side
        int totalElements = curr_inBytes / sizeof(float);
        int rows = (totalElements + options.width - 1) / options.width;
        int in_cols = options.width;

        std::vector<float> h_in(rows * in_cols, 0);
        memcpy(h_in.data(), recv_buffer.data(), curr_inBytes);

        if (verbose_mode) {
            std::cout << "First 10 elements of h_in:" << std::endl;
            for (size_t i = 0; i < std::min(h_in.size(), static_cast<size_t>(10)); ++i) {
                std::cout << h_in[i] << " ";
            }
            std::cout << std::endl << std::endl;
        }

        // Copy input matrix to GPU
        CUDA_CALL(hipMalloc(&d_in, rows * in_cols * sizeof(float)));
        if (verbose_mode) {
            std::cout << "\t Input matrix dimension, (#columns)x(#rows): " << in_cols << "x" << rows << std::endl;
        }
        CUDA_CALL(hipMemcpy(d_in, h_in.data(), rows * in_cols * sizeof(float), hipMemcpyHostToDevice));

        // Set the random matrix d_rand on the GPU. d_rand has @var options.width rows.
        int out_cols = std::ceil(options.width * options.rate);
        int rand_elements = in_cols * out_cols;
        CUDA_CALL(hipMalloc(&d_rand, rand_elements * sizeof(float)));

        int threadsPerBlock = 256;
        int numBlocks = (rand_elements + threadsPerBlock - 1) / threadsPerBlock;
        if (verbose_mode) {
            std::cout << "\t Random matrix dimension, (#columns)x(#rows): " << out_cols << "x" << options.width << std::endl;
        }
        generateRandomMatrix<<<numBlocks, threadsPerBlock>>>(d_rand, options.width, out_cols, time(NULL));
        CUDA_CALL(hipDeviceSynchronize());

        // Process matrix multiplication
        CUDA_CALL(hipMalloc(&d_out, rows * out_cols * sizeof(float)));
        matrixProcess(d_in, d_rand, d_out, rows, in_cols, out_cols);

        CUDA_CALL(hipDeviceSynchronize());

        // Copy the result matrix back to host
        std::vector<float> h_out(rows * out_cols, 0);
        CUDA_CALL(hipMemcpy(h_out.data(), d_out, rows * out_cols * sizeof(float), hipMemcpyDeviceToHost));

        if (verbose_mode) {
            std::vector<float> h_rand(rand_elements, 0);
            CUDA_CALL(hipMemcpy(h_rand.data(), d_rand, rand_elements * sizeof(float), hipMemcpyDeviceToHost));
    
            std::cout << "First 10 elements of h_out:" << std::endl;
            for (size_t i = 0; i < std::min(h_out.size(), static_cast<size_t>(10)); ++i) {
                std::cout << h_out[i] << " ";
            }
            std::cout << std::endl << std::endl;

            std::vector<float> h_out_ref(rows * out_cols, 0);
            cpuMatrixMultiply(h_in, h_rand, h_out_ref, rows, in_cols, out_cols);
            std::cout << "\nFirst 10 elements of CPU computed matrix multiplication result:" << std::endl;
            for (size_t i = 0; i < std::min(h_out_ref.size(), static_cast<size_t>(10)); ++i) {
                std::cout << h_out_ref[i] << " ";
            }
            std::cout << std::endl << std::endl;
        }

        zmq::message_t message(h_out.data(), h_out.size() * sizeof(float));   // remember to * sizeof(float)!!!
        if (verbose_mode) {
            std::cout <<"\t Output matrix dimension, (#columns)x(#rows): " << out_cols << "x" << rows << std::endl;
        }

        res = sender.send(message, zmq::send_flags::dontwait);   // zmq::send_flags::dontwait is non-blocking mode
        if (!res) {
            std::cerr << "Error: ZeroMQ send failed!" << std::endl;
        }
        totalOutBytes += res.value();
        if (verbose_mode) {
            std::cout << "Sent [" << res.value() << "] bytes via ZeroMQ socket.\n" << std::endl;
        }

        CUDA_CALL(hipFree(d_in));
        CUDA_CALL(hipFree(d_rand));
        CUDA_CALL(hipFree(d_out));

    }

    return 0;
}
