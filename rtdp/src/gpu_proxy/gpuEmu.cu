#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------
// This program will:
//  1. Listen to the imcoming traffic from a TCP port via ZMQ.
//  2. Copy the incoming data to GPU and do some fake calculation on GPU.
//  3. Copy the results from GPU to CPU and send them out via ZMQ.
//
//---------------------------------------------------------------------------

#include <iostream>
#include <thread>
#include <string>
#include <chrono>
#include <vector>
#include <cmath>
#include <atomic>
#include <unistd.h> // For getpid()
#include <iomanip>
#include <ctime>

#include <zmq.hpp>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>


// ZMQ port settings
constexpr const int ZMQ_IN_PORT = 55555;
constexpr const int ZMQ_OUT_PORT = 55556;

//.........................................................................
// Matrix multiplication setup

constexpr int MATRIX_IN_COLUMN_WIDTH = 2048;
constexpr float MATRIX_OUT_REDUCE_RATE = 1.0; // Define reduction rate

// CUDA error check
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(x) << std::endl; \
    exit(EXIT_FAILURE); }} while(0)

// CUBLAS error check
#define CUBLAS_CALL(call) \
do { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "CUBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// GPU kernel function
__global__ void generateRandomMatrix(float* d_rand, int rows, int cols, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = rows * cols;
    if (idx < totalElements) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        d_rand[idx] = hiprand_uniform(&state);
    }
}

// Excute on GPU
void matrixProcess(float* d_A, float* d_B, float* d_C, int rows, int in_cols, int out_cols) {
    hipblasHandle_t handle;
    CUBLAS_CALL(hipblasCreate(&handle));
    const float alpha = 1.0f, beta = 0.0f;

    // Perform matrix multiplication: C = A * B
    CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            out_cols, rows, in_cols,
                            &alpha, d_B, out_cols,
                            d_A, in_cols,
                            &beta, d_C, out_cols));

    CUBLAS_CALL(hipblasDestroy(handle));
}

// Execute on CPU.
void cpuMatrixMultiply(
    const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& C,
    int rows, int in_cols, int out_cols) {
        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < out_cols; ++j) {
                float sum = 0.0f;
                for (int k = 0; k < in_cols; ++k) {
                    sum += A[i * in_cols + k] * B[k * out_cols + j];
                }
                C[i * out_cols + j] = sum;
            }
        }
}
//.........................................................................


//.........................................................................
// CommandLine inputs
class CommandLineOptions {
public:
    int recv_port = ZMQ_IN_PORT;
    int send_port = ZMQ_OUT_PORT;
    double rate = MATRIX_OUT_REDUCE_RATE;
    int width = MATRIX_IN_COLUMN_WIDTH; // Default matrix column size

    std::string in_ip = "127.0.0.1";  // Default out IP is local
    bool verbose = false;

    static CommandLineOptions Parse(int argc, char* argv[]) {
        CommandLineOptions options;
        for (int i = 1; i < argc; ++i) {
            std::string arg = argv[i];
            if (arg == "-h" || arg == "--help") {
                PrintUsage();
                exit(0);
            } else if (arg == "-i" || arg == "--in-ip" && i + 1 < argc) {
                options.in_ip = argv[++i];
            } else if (arg == "-p" ||arg == "--in-port") {
                options.recv_port = std::stoi(argv[++i]);
            } else if (arg == "-r" || arg == "--out-port") {
                options.send_port = std::stoi(argv[++i]);
            } else if (arg == "--rate") {
                if (i + 1 < argc) {
                    options.rate = std::stod(argv[++i]);
                }
            } else if (arg == "-w" || arg == "--width") {
                if (i + 1 < argc) {
                    options.width = std::stoi(argv[++i]);
                }
            }
            /* For future extension
            else if (arg == "-s" || arg == "--sqlfile") {
                if (i + 1 < argc) {
                    options.sqliteFilename = argv[++i];
                }
            } else if (arg == "--tc") {
                options.useTensorCores = true;
            } */
            else if (arg == "-v" || arg == "--verbose") {
                options.verbose = true;
            }
        }
        return options;
    }

    static void PrintUsage() {
        std::cout << "\n"
                  << "Usage: gpu_emu [--in-port] [-a|--out-ip] [--out-port]\n"
                  << "\n"
                  << "-h, --help     Print this help statement\n"
                  << "-i, --in-ip    <IPv4_Address> The IP address ZMQ subscribe from (default is localhost)\n"
                  << "-p, --in-port  <incoming port> Set ZMQ port to subscribe from (default is 55555)\n"
                  << "-r, --out-port <outgoing port> Set ZMQ port to publish to (default is 55556)\n"
                  << "    --rate     Control the ratio of output/input volume (default is 1.0)\n"
                  << "-w, --width    Set the GPU input matrix column size (default is 2048)\n"
                //   << "-s, --sqlfile  <file> Specify the SQL rate logger file\n"
                //   << "    --tc       Use GPU Tensor Cores instead of FP units\n"
                  << "-v, --verbose  Enable the verbose mode (default is false)\n"

                  << "\n"
                  << "This is a GPU Proxy\n"
                  << "It takes input from a ZMQ SUB IPv4 address and builds a matrix, sends to GPU and\n"
                  << "do matrix multiplication on the GPU. After that, it copies the result back to CPU\n"
                  << "and PUB to another ZMQ TCP port.\n"
                  << "\n"
                //   << "If --sqlfile is used, it specifies a SQLite rate logger.\n"
                  << "\n";
    }
};
//.........................................................................


//.........................................................................
// The monitoring thread
std::atomic<size_t> totalInBytes{0};
std::atomic<size_t> totalOutBytes{0};

static std::string utc_now_hms_ms() {
    using namespace std::chrono;
    auto now = system_clock::now();
    auto tt  = system_clock::to_time_t(now);
    auto ms  = duration_cast<milliseconds>(now.time_since_epoch()) % 1000;
    std::tm tm{};
#if defined(_WIN32)
    gmtime_s(&tm, &tt);
#else
    gmtime_r(&tt, &tm);
#endif
    std::ostringstream oss;
    oss << '[' << std::put_time(&tm, "%Y-%m-%d %H:%M:%S")
        << '.' << std::setfill('0') << std::setw(3) << ms.count()
        << " UTC]";
    return oss.str();
}


void monitorTraffic(std::atomic<size_t>* inBytes, std::atomic<size_t>* outBytes) {
    using namespace std::chrono;
    constexpr int sleep_in_seconds = 2;
    while (true) {
        size_t prevIn  = inBytes->load(std::memory_order_relaxed);
        size_t prevOut = outBytes->load(std::memory_order_relaxed);
        auto t0 = steady_clock::now();
        std::this_thread::sleep_for(seconds(sleep_in_seconds));
        auto t1 = steady_clock::now();
        double elapsed = duration_cast<duration<double>>(t1 - t0).count();
        size_t curIn  = inBytes->load(std::memory_order_relaxed);
        size_t curOut = outBytes->load(std::memory_order_relaxed);

        double inRate_Gbps  = 8.0e-9 * double(curIn  - prevIn)  / elapsed;
        double outRate_Gbps = 8.0e-9 * double(curOut - prevOut) / elapsed;

        if (inRate_Gbps == 0.0 && outRate_Gbps == 0.0)
            continue;       // skip low rate logging
        std::cout << utc_now_hms_ms()
                  << "  Incoming: [" << inRate_Gbps  << " Gbps], total [" << curIn/1.0e6 << " MB] "
                  << "; Outgoing: [" << outRate_Gbps << " Gbps], total [" << curOut/1.0e6 << " MB] "
                  << std::endl;
    }
}
//.........................................................................


//.........................................................................
// Main
int main(int narg, char *argv[]){

    // Parse command options (will print help and exit if help is asked for)
    CommandLineOptions options = CommandLineOptions::Parse(narg, argv);

    // Enable the verbose mode if the cmd flag is provided
    bool verbose_mode = options.verbose;

    //............................................
    // Setup network communication via zmq
    zmq::context_t context(1);

    // Receiving socket. Subscribe from an extrnal/upstream IPv4 address.
    zmq::socket_t receiver(context, ZMQ_SUB);
    std::string recv_addr = "tcp://"+ options.in_ip + ":" + std::to_string(options.recv_port);
    try {
        receiver.connect(recv_addr.c_str());
        receiver.set(zmq::sockopt::subscribe, "");    // subscribe to all topics
        std::cout << "SUB - ZeroMQ subscribed to: " << recv_addr << "\n";
    }  catch (const zmq::error_t& e) {
        std::cout << "Error: Failed to bind to the receiving address [" << recv_addr << "]:" << e.what() << "\n";
        return 1;
    }

    // Sending socket. Publish to a port on localhost.
    zmq::socket_t sender(context, ZMQ_PUB);
    std::string send_addr = "tcp://*:" + std::to_string(options.send_port);

    try {
        sender.bind(send_addr.c_str());
        std::cout << "PUB - ZeroMQ publishing on: " << send_addr << "\n";
    } catch (const zmq::error_t& e) {
        std::cout << "Error: Failed to bind PUB [" << send_addr << "]:" << e.what() << "\n";
        return 1;
    }
    //............................................

    std::cout << "\nWaiting for data ...\n" << std::endl;

    std::thread monitor_thread(monitorTraffic, &totalInBytes, &totalOutBytes);
    monitor_thread.detach();   // Start the rate monitoring thread

    while (true) {
        zmq::message_t recv_buffer;
        auto res = receiver.recv(recv_buffer, zmq::recv_flags::none);
        if (!res) {
            std::cerr << "Error: ZeroMQ receive failed!" << std::endl;
        }

        if (verbose_mode) {
            std::cout << "Received [" << res.value() << "] bytes from ZeroMQ socket." << std::endl;
        }

        size_t curr_inBytes = recv_buffer.size();
        totalInBytes += curr_inBytes;
        if( curr_inBytes == 0 ) {
            if (verbose_mode) {
                std::cout << "  (skipping empty buffer)" << std::endl;
            }
            continue;
        }

        float *d_in, *d_rand, *d_out;
        // Setup the input matrix A [rows * in_columns (default as 2048)] on the CPU side
        int totalElements = curr_inBytes / sizeof(float);
        int rows = (totalElements + options.width - 1) / options.width;
        int in_cols = options.width;

        std::vector<float> h_in(rows * in_cols, 0);
        memcpy(h_in.data(), recv_buffer.data(), curr_inBytes);

        if (verbose_mode) {
            std::cout << "First 10 elements of h_in:" << std::endl;
            for (size_t i = 0; i < std::min(h_in.size(), static_cast<size_t>(10)); ++i) {
                std::cout << h_in[i] << " ";
            }
            std::cout << std::endl << std::endl;
        }

        // Copy input matrix to GPU
        CUDA_CALL(hipMalloc(&d_in, rows * in_cols * sizeof(float)));
        if (verbose_mode) {
            std::cout << "\t Input matrix dimension, (#columns)x(#rows): " << in_cols << "x" << rows << std::endl;
        }
        CUDA_CALL(hipMemcpy(d_in, h_in.data(), rows * in_cols * sizeof(float), hipMemcpyHostToDevice));

        // Set the random matrix d_rand on the GPU. d_rand has @var options.width rows.
        int out_cols = std::ceil(options.width * options.rate);
        int rand_elements = in_cols * out_cols;
        CUDA_CALL(hipMalloc(&d_rand, rand_elements * sizeof(float)));

        int threadsPerBlock = 256;
        int numBlocks = (rand_elements + threadsPerBlock - 1) / threadsPerBlock;
        if (verbose_mode) {
            std::cout << "\t Random matrix dimension, (#columns)x(#rows): " << out_cols << "x" << options.width << std::endl;
        }
        generateRandomMatrix<<<numBlocks, threadsPerBlock>>>(d_rand, options.width, out_cols, time(NULL));
        CUDA_CALL(hipDeviceSynchronize());

        // Process matrix multiplication
        CUDA_CALL(hipMalloc(&d_out, rows * out_cols * sizeof(float)));
        matrixProcess(d_in, d_rand, d_out, rows, in_cols, out_cols);

        CUDA_CALL(hipDeviceSynchronize());

        // Copy the result matrix back to host
        std::vector<float> h_out(rows * out_cols, 0);
        CUDA_CALL(hipMemcpy(h_out.data(), d_out, rows * out_cols * sizeof(float), hipMemcpyDeviceToHost));

        if (verbose_mode) {
            std::vector<float> h_rand(rand_elements, 0);
            CUDA_CALL(hipMemcpy(h_rand.data(), d_rand, rand_elements * sizeof(float), hipMemcpyDeviceToHost));

            std::cout << "First 10 elements of h_out:" << std::endl;
            for (size_t i = 0; i < std::min(h_out.size(), static_cast<size_t>(10)); ++i) {
                std::cout << h_out[i] << " ";
            }
            std::cout << std::endl << std::endl;

            std::vector<float> h_out_ref(rows * out_cols, 0);
            cpuMatrixMultiply(h_in, h_rand, h_out_ref, rows, in_cols, out_cols);
            std::cout << "\nFirst 10 elements of CPU computed matrix multiplication result:" << std::endl;
            for (size_t i = 0; i < std::min(h_out_ref.size(), static_cast<size_t>(10)); ++i) {
                std::cout << h_out_ref[i] << " ";
            }
            std::cout << std::endl << std::endl;
        }

        zmq::message_t message(h_out.data(), h_out.size() * sizeof(float));   // remember to * sizeof(float)!!!
        if (verbose_mode) {
            std::cout <<"\t Output matrix dimension, (#columns)x(#rows): " << out_cols << "x" << rows << std::endl;
        }

        res = sender.send(message, zmq::send_flags::dontwait);   // zmq::send_flags::dontwait is non-blocking mode
        if (!res) {
            std::cerr << "Error: ZeroMQ send failed!" << std::endl;
        }
        totalOutBytes += res.value();
        if (verbose_mode) {
            std::cout << "Sent [" << res.value() << "] bytes via ZeroMQ socket.\n" << std::endl;
        }

        CUDA_CALL(hipFree(d_in));
        CUDA_CALL(hipFree(d_rand));
        CUDA_CALL(hipFree(d_out));
    }

    return 0;
}
